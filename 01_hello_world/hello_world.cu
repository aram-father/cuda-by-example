
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

__global__ void kernel_add(int src_0, int src_1, int* dst)
{
  *dst = src_0 + src_1;
}

int main(int argc, char** argv)
{
  if (argc != 3)
  {
    std::cout << "Uasge: 01_hello_world number_0 number_1" << std::endl;
    exit(-1);
  }

  hipError_t error;

  int src_0 = std::atoi(argv[1]);
  int src_1 = std::atoi(argv[2]);
  int *pdst, dst;

  error = hipMalloc(reinterpret_cast<void**>(&pdst), sizeof(int));
  if (error)
  {
    std::cout << hipGetErrorString(error) << std::endl;
    exit(-1);
  }

  kernel_add<<<1,1>>>(src_0, src_1, pdst);

  error = hipMemcpy(&dst, pdst, sizeof(int), hipMemcpyDeviceToHost);
  if (error)
  {
    std::cout << hipGetErrorString(error) << std::endl;
    exit(-1);
  }

  std::cout << src_0 << "+" << src_1 << "=" << dst << std::endl;

  hipFree(pdst);

  return 0;
}