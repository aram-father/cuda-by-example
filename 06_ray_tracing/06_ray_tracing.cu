/**
 * @file 06_ray_tracing.cu
 * @author Wonseok Lee (aram_father@naver.com)
 * @brief Main for ray tracing example
 * @version 0.1
 * @date 2021-05-03
 * 
 * @copyright Copyright (c) 2021
 * 
 */
#include <ctime>
#include <string>
#include <cstdlib>
#include <iostream>

#include "bitmap.hpp"
#include "sphere.cuh"
#include "ray_tracing.cuh"

static void HandleError(hipError_t error, const char *file, int line)
{
  if (error != hipSuccess)
  {
    std::cout << hipGetErrorString(error) << " @ " << line << " of " << file << std::endl;
    exit(-1);
  }
}

#define HANDLE_ERROR(error) (HandleError((error), __FILE__, __LINE__))

int main(int argc, char **argv)
{
  int number_of_spheres;

  try
  {
    if (argc != 2 || !(number_of_spheres = std::stoi(argv[1])) || number_of_spheres >= 1024)
    {
      throw std::string("Usage: ./06_ray_tracing NUMBER_OF_SPHERES (0,1024]");
    }
  }
  catch (std::string usage_exception)
  {
    std::cout << usage_exception << std::endl;
    exit(-1);
  }

  Sphere* p_spheres_host = new Sphere[number_of_spheres];
  if (!p_spheres_host)
  {
    std::cout << "host heap allocation failed" << std::endl;
    exit(-1);
  }

  std::srand(std::time(0));
  for (int sphere_idx = 0; sphere_idx < number_of_spheres; ++sphere_idx)
  {
    p_spheres_host[sphere_idx].Initialize();
  }

  float elapsed_time;
  hipEvent_t start, end;

  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&end));

  const std::uint32_t kNRow = MAX_X * 2;
  const std::uint32_t kNCol = MAX_Y * 2;
  
  Bitmap bmp(kNRow, kNCol);
  
  Sphere* p_spheres_dev;
  std::uint8_t* p_image_dev;

  dim3 grid_dim(kNRow / 16, kNCol / 16, 1);
  dim3 block_dim(16, 16, 1);

  // Global memory version
  HANDLE_ERROR(hipEventRecord(start, 0));
  
  HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&p_spheres_dev), sizeof(Sphere) * number_of_spheres));
  HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&p_image_dev), sizeof(std::uint8_t) * kNRow * kNCol * 4));
  HANDLE_ERROR(hipMemcpy(p_spheres_dev, p_spheres_host, sizeof(Sphere) * number_of_spheres, hipMemcpyHostToDevice));

  DoRayTracingUsingGlobalMemory<<<grid_dim, block_dim>>>(number_of_spheres, p_spheres_dev, p_image_dev);

  HANDLE_ERROR(hipMemcpy(bmp.image().data, p_image_dev, sizeof(std::uint8_t) * kNRow * kNCol * 4, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(p_spheres_dev));
  HANDLE_ERROR(hipFree(p_image_dev));
  
  HANDLE_ERROR(hipEventRecord(end, 0));
  HANDLE_ERROR(hipEventSynchronize(end));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, end));
  std::cout << "Global memory version elapsed time: " << elapsed_time << std::endl;
  bmp.ShowAndWait();

  // Constant memory version
  HANDLE_ERROR(hipEventRecord(start, 0));

  HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&p_image_dev), sizeof(std::uint8_t) * kNRow * kNCol * 4));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(kSpheresDev), p_spheres_host, sizeof(Sphere) * number_of_spheres));

  DoRayTracingUsingConstantMemory<<<grid_dim, block_dim>>>(number_of_spheres, p_image_dev);

  HANDLE_ERROR(hipMemcpy(bmp.image().data, p_image_dev, sizeof(std::uint8_t) * kNRow * kNCol * 4, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(p_image_dev));
  
  HANDLE_ERROR(hipEventRecord(end, 0));
  HANDLE_ERROR(hipEventSynchronize(end));
  HANDLE_ERROR(hipEventElapsedTime(&elapsed_time, start, end));
  std::cout << "Constant memory version elapsed time: " << elapsed_time << std::endl;
  bmp.ShowAndWait();

  // Common termination
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(end));

  free(p_spheres_host);

  return 0;
}