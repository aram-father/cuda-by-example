/**
 * @file main.cc
 * @author your name (you@domain.com)
 * @brief Main
 * @version 0.1
 * @date 2021-03-08
 *
 * @copyright Copyright (c) 2021
 *
 */
#include <cstdlib>
#include <iostream>

#include "bitmap.hpp"
#include "julia_set.cuh"

int main(int argc, char** argv)
{
  if (argc != 3)
  {
    std::cout << "Usage: ./main dim_x dim_y" << std::endl;
    exit(-1);
  }

  int nrow = std::atoi(argv[1]);
  int ncol = std::atoi(argv[2]);

  std::uint8_t* p_image_dev;
  Bitmap bmp(nrow, ncol);

  hipError_t error;
  error = hipMalloc(reinterpret_cast<void**>(&p_image_dev), nrow * ncol * 4);
  if (error)
  {
    std::cout << hipGetErrorString(error) << std::endl;
    exit(-1);
  }

  dim3 grid(nrow, ncol);
  kernel<<<grid, 1>>>(p_image_dev);

  error = hipMemcpy(bmp.image().data, p_image_dev, nrow * ncol * 4, hipMemcpyDeviceToHost);
  if (error)
  {
    std::cout << hipGetErrorString(error) << std::endl;
  }

  error = hipFree(p_image_dev);
  if (error)
  {
    std::cout << hipGetErrorString(error) << std::endl;
    exit(-1);
  }

  bmp.ShowAndWait("test");

  return 0;
}